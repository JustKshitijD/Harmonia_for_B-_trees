#include "hip/hip_runtime.h"
# include <bits/stdc++.h>
# define MAX 1000000000
using namespace std;

__device__ int k1_gpu=364; __device__ int k2_gpu=121;   // k1= 3^0+...+3^5 = 364= sizeof(nodes)
                          //  k2=364-3^5= sizeof(prefix_sum)
__device__ int m_gpu=2;                  // 1 key + 4 data items

float avg_tree_size=0.0;
int T_count=0;

__device__ int GS=3; __device__ int no_of_queries_in_warp=6;
int GS_cpu=3; int no_of_queries_in_warp_cpu=6;
int fanout=4; __device__ int fanout_gpu=8;             // so, each node can have max fanout-1 keys and fanout children
int nk=3;
__device__ int nk_gpu=3;


__device__ typedef struct Node
{
  int* key;      // keys are sorted
  int** data;
}node_gpu;

__device__ node_gpu** nodes_gpu;
__device__ int* prefix_sum_gpu;

int k1=364; int k2=121;

int m=2;
int sum_of_den=0;

bool flag=false;
int incremented_index=-1;


typedef struct Node_gpu
{
  int* key;      // keys are sorted
  int** data;
}node;


node** nodes;
int* prefix_sum;

int lb(int* arr, int N, int X)                     // binary search to find element, and give found index or index of next-highest element
{
	int mid;

	// Initialise starting index and
	// ending index
	int low = 0;
	int high = N;

	// Till low is less than high
	while (low < high) {
		mid = low + (high - low) / 2;

		// If X is less than or equal
		// to arr[mid], then find in
		// left subarray
		if (X <= arr[mid]) {
			high = mid;
		}

		// If X is greater arr[mid]
		// then find in right subarray
		else {
			low = mid + 1;
		}
	}

	// Return the lb index
	return low;
}


void init(node* nn)
{
  nn->key=(int*)malloc(sizeof(int)*nk);                // init each node with nk keys
  nn->data=(int**)malloc(sizeof(int*)*nk);             // and nk data arrays, each having m integers
  for(int i=0;i<nk;i++)
  {
    nn->data[i]=(int*)malloc(sizeof(int)*m);
  }

  for(int i=0;i<nk;i++)
    nn->key[i]=MAX;

    for(int i=0;i<nk;i++)
    {
      for(int j=0;j<m;j++)
      {
        nn->data[i][j]=MAX;
      }
    }
}

__device__ void init_gpu(node_gpu* nn)
{
    nn->key=(int*)malloc(sizeof(int)*nk_gpu);                // init each node with 3 keys
  nn->data=(int**)malloc(sizeof(int*)*nk_gpu);             // and 3 data arrays, each having m integers
  for(int i=0;i<nk_gpu;i++)
  {
    nn->data[i]=(int*)malloc(sizeof(int)*m_gpu);
  }

  for(int i=0;i<nk_gpu;i++)
    nn->key[i]=MAX;

    for(int i=0;i<nk_gpu;i++)
    {
      for(int j=0;j<m_gpu;j++)
      {
        nn->data[i][j]=MAX;
      }
    }
}

__global__ void iii()                                 // initializing nodes array in GPU
{
  nodes_gpu=(node_gpu**)malloc(sizeof(node_gpu*)*k1_gpu);
  for(int i=0;i<k1_gpu;i++)
    nodes_gpu[i]=NULL;
}

__global__ void cpy_key(int* key,int *i2)             // copy key to a node in GPU
{
    int i=*i2;
    printf("In cpy_key, i: %d\n",i);
    nodes_gpu[i]=(node_gpu*)malloc(sizeof(node_gpu));
    init_gpu(nodes_gpu[i]);

    for(int h=0;h<nk_gpu;h++)
    {
        nodes_gpu[i]->key[h]=key[h];
    }

}

__global__ void cpy_data(int* data,int* i2, int* ii2)    // ith node in nodes[]; iith key in that node
{
    int i=*i2; int ii=*ii2;
    nodes_gpu[i]->data[ii]=data;
    for(int h=0;h<m_gpu;h++)
    {
        (nodes_gpu[i]->data)[ii][h]=data[h];
    }
}

__global__ void print()
{
  printf("In print(), k1_gpu: %d\n",k1_gpu);
    for(int i=0;i<k1_gpu && nodes_gpu[i]!=NULL;i++)
    {
        printf("nodes_gpu[%d]->key: \n",i);
        for(int j=0;j<nk_gpu;j++)
        {
            printf("%d ",(nodes_gpu[i]->key)[j]);
        }
        printf("\n");

        printf("nodes_gpu[%d]->data: \n",i);
        for(int j=0;j<nk_gpu;j++)
        {
            for(int kk=0;kk<m_gpu;kk++)
            {
              printf("%d ",(nodes_gpu[i]->data)[j][kk]);
            }
            printf("\n");
        }
        printf("\n");
    }
}


int search(int key)                               // search for index of node in nodes[], having the given key
{
  int i=0; //node* ptr; ptr=nodes[0];
  int j=0;

  while(true)
  {
    //ptr=nodes[i];
    j=i;
    if(nodes[i]==NULL)                     // no node inserted yet
      return 0;
    int* key_arr=nodes[i]->key;
    int ind=lb(key_arr,nk,key);
    cout<<"i: "<<i<<"; ind: "<<ind<<" ; prefix_sum[i]: "<<prefix_sum[i]<<endl;
    if(prefix_sum[i]==MAX)                  // reached a leaf
      break;
    i=prefix_sum[i]+ind;
    if(key_arr[ind]==key)
      i++;
  }

  int* key_arr=nodes[j]->key;
  int ind=lb(key_arr,nk,key);
  //cout<<"SEARCH"<<endl;
  if(key_arr[ind]==key)
  {
    int** dd=nodes[j]->data;
    for(int r=0;r<m;r++)
        cout<<dd[ind][r]<<" ";
    cout<<endl;
  }

  return j;
}

void range_query(int k1,int k2)                   // scan leaves from first key to last key
{
  int j=search(k1);
  int* key_arr=nodes[j]->key;
  int ind=lb(key_arr,nk,k1);
  //cout<<"SEARCH"<<endl;
  cout<<"range_query: "<<endl;
  if(key_arr[ind]==k1)
  {
    cout<<"In"<<endl;
    ind++;
    int** dd=nodes[j]->data;

    for(;ind<nk;ind++)
    {
      if(key_arr[ind]==MAX)
        break;
      for(int r=0;r<m;r++)
          cout<<dd[ind][r]<<" ";
     }
    cout<<endl;
  }

  j++; bool f2=false;

  while(nodes[j]!=NULL && f2==false)
  {
    int* key_arr=nodes[j]->key;
    int** dd=nodes[j]->data;
    int ind=0;
    for(;ind<nk ;ind++)
    {
      if(key_arr[ind]==MAX)
        break;
      if(dd[ind][0]>k2)
      {
        f2=true;
        break;
      }
      for(int r=0;r<m;r++)
          cout<<dd[ind][r]<<" ";
      cout<<endl;
    }
    j++;
  }
}

void put_in_middle(node* nn,int pos, int key, int* dd)   // put key and its data dd at position pos in nn
{
  int arr[nk]; int dt[nk][m];
  for(int i=0;i<nk;i++)
    arr[i]=nn->key[i];

  for(int i=0;i<nk;i++)
  {
    for(int j=0;j<m;j++)
    {
      dt[i][j]=nn->data[i][j];
    }
  }

  nn->key[pos]=key;
  for(int i=pos+1;i<nk;i++)
    nn->key[i]=arr[i-1];

  for(int j=0;j<m;j++)
    nn->data[pos][j]=dd[j];

    for(int i=pos+1;i<nk;i++)
    {
      for(int j=0;j<m;j++)
      {
        nn->data[i][j]=dt[i-1][j];
      }
    }
}

void put_in_nodes(int ind,node* nn2)            // put nn2 in nodes at ind position
{
  node* g=nodes[ind]; nodes[ind]=nn2; int i=ind+1;
  for(i=ind+1;i<k1 && nodes[i]!=NULL;i++)
  {
    node* h=nodes[i];
    nodes[i]=g;
    g=h;
  }
  nodes[i]=g;
}

void put_in_array(int* arr, int sz, int x, int ind)      // put value x at index ind in array arr. sz is size of array
{
  int g=arr[ind]; arr[ind]=x; int i=ind+1;
  for(i=ind+1;i<k1 && arr[i]!=MAX;i++)
  {
    int h=arr[i];
    arr[i]=g;
    g=h;
  }
  arr[i]=g;
}

int parent(int ind)                           // give index of parent of node ind
{
  if(prefix_sum[0]==MAX || ind==0)                      // no element in prefix_sum
    return -1;

  int p=lb(prefix_sum,k2,ind);
  //cout<<"pp: "<<p<<"; prefix_sum[p]: "<<prefix_sum[p]<<endl;
  if(prefix_sum[p]!=ind)
    p--;
  return p;
}


int insert_internal(node* nn,int ind,int key,int* dd,int orig)    // insert key in node* nn
{
  cout<<"start ind: "<<ind<<"; start key: "<<key<<endl;
  cout<<"Keys of start node nn: "<<endl;
  for(int i=0;i<nk;i++)
    cout<<nn->key[i]<<" ";
  cout<<endl;

  int pos=lb(nn->key,nk,key);
  if(nn->key[nk-1]==MAX)                  // space there in node
  {
      cout<<"Inserting normally"<<endl;
      put_in_middle(nn,pos,key,dd);
      return 0;

  }
  else                                    // node is full
  {
    cout<<"Node is full"<<endl;
    for(int i=0;i<nk;i++)
    {
      cout<<nn->key[i]<<"# "<<endl;
    }
    cout<<"key: "<<key<<endl;

    int pos=lb(nn->key,nk,key); node* nn2; int* dd_mid;
    dd_mid=(int*)malloc(m*sizeof(int));
   int middle_element;  // middle_element goes to top next time
   cout<<"pos: "<<pos<<endl;

    if(prefix_sum[ind]==MAX)               // leaf
    {
      int mid=(nk+1)/2;
      cout<<"mid: "<<mid<<endl;

      if(pos<mid)
      {
        nn2=(node*)malloc(sizeof(node));
        init(nn2); middle_element=nn->key[mid-1]; // mid-1 goes to top
        for(int j=0;j<m;j++)
          dd_mid[j]=nn->data[mid-1][j];

        for(int i=mid-1;i<nk;i++)
        {
          nn2->key[i-(mid-1)]=nn->key[i];
        }

        for(int i=mid-1;i<nk;i++)
        {
          for(int j=0;j<m;j++)
          {
            nn2->data[i-(mid-1)][j]=nn->data[i][j];
          }
        }

        for(int i=mid-1;i<nk;i++)
        {
          nn->key[i]=MAX;
        }
        for(int i=mid-1;i<nk;i++)
        {
          for(int j=0;j<m;j++)
          {
            nn->data[i][j]=MAX;
          }
        }

        put_in_middle(nn,pos,key,dd);

      }
      else
      {
        if(pos==mid)
        {
          middle_element=key;
          for(int j=0;j<m;j++)
            dd_mid[j]=dd[j];
        }
        else
        {
          middle_element=nn->key[mid];
          cout<<"middle_element: "<<middle_element<<endl;
          for(int j=0;j<m;j++)
          {
            cout<<"j: "<<j<<endl;
            dd_mid[j]=nn->data[mid][j];
          }
        }

        nn2=(node*)malloc(sizeof(node));
        init(nn2);

        for(int i=mid;i<nk;i++)
        {
          nn2->key[i-mid]=nn->key[i];
        }
        for(int i=mid;i<nk;i++)
        {
          for(int j=0;j<m;j++)
          {
            nn2->data[i-mid][j]=nn->data[i][j];
          }
        }
        put_in_middle(nn2,pos-mid,key,dd);

        cout<<"mid2: "<<mid<<endl;
        for(int i=mid;i<nk;i++)
        {
          cout<<"i: "<<i<<endl;
          nn->key[i]=MAX;
        }

        for(int i=mid;i<nk;i++)
        {
          for(int j=0;j<m;j++)
          {
            nn->data[i][j]=MAX;
          }
        }
      }

    }
    else
    {
      int mid=(nk+1)/2;

      if(pos<mid)
      {
        nn2=(node*)malloc(sizeof(node));
        init(nn2); middle_element=nn->key[mid-1]; // mid-1 goes to top
        for(int j=0;j<m;j++)
          dd_mid[j]=nn->data[mid-1][j];

        for(int i=mid;i<nk;i++)                  // leaving mid-1
        {
          nn2->key[i-(mid)]=nn->key[i];
        }


        for(int i=mid;i<nk;i++)
        {
          for(int j=0;j<m;j++)
          {
            nn2->data[i-(mid)][j]=nn->data[i][j];
          }
        }

        for(int i=mid-1;i<nk;i++)
        {
          nn->key[i]=MAX;
        }
        for(int i=mid-1;i<nk;i++)
        {
          for(int j=0;j<m;j++)
          {
            nn->data[i][j]=MAX;
          }
        }

        put_in_middle(nn,pos,key,dd);

      }
      else
      {
        if(pos==mid)
        {
          middle_element=key;
          for(int j=0;j<m;j++)
            dd_mid[j]=dd[j];
          nn2=(node*)malloc(sizeof(node));
          init(nn2);

          for(int i=mid;i<nk;i++)
          {
            nn2->key[i-mid]=nn->key[i];
          }
          for(int i=mid;i<nk;i++)
          {
            for(int j=0;j<m;j++)
            {
              nn2->data[i-mid][j]=nn->data[i][j];
            }
          }

        }
        else
        {
          middle_element=nn->key[mid];
          for(int j=0;j<m;j++)
            dd_mid[j]=nn->data[mid][j];
          nn2=(node*)malloc(sizeof(node));
          init(nn2);

          for(int i=mid+1;i<nk;i++)
          {
            nn2->key[i-(mid+1)]=nn->key[i];
          }
          for(int i=mid+1;i<nk;i++)
          {
            for(int j=0;j<m;j++)
            {
              nn2->data[i-(mid+1)][j]=nn->data[i][j];
            }
          }
          put_in_middle(nn2,pos-(mid+1),key,dd);
        }

        for(int i=mid;i<nk;i++)
        {
          nn->key[i]=MAX;
        }
        for(int i=mid;i<nk;i++)
        {
          for(int j=0;j<m;j++)
          {
            nn->data[i][j]=MAX;
          }
        }
      }
    }

    put_in_nodes(ind+1,nn2);

    if(prefix_sum[ind]==MAX)               // leaf
    {
      int xx=0; int p=parent(ind);
      if(p==-1 || nodes[p]==NULL)     // leaf and parent is null means first node to be filled, and now, prefix_sum will get its first entry
      {
        cout<<endl;
        cout<<"Leaf with parent NULL:- ind: "<<ind<<"; p: "<<p<<endl;
        prefix_sum[0]=1;
        node* nn3; nn3=(node*)malloc(sizeof(node));
        init(nn3);
        //cout<<"middle_element: "<<middle_element<<endl;
        put_in_middle(nn3,0,middle_element,dd_mid);
        //cout<<"middle_element: "<<middle_element<<endl;
        put_in_nodes(0,nn3);
        return 0;
      }
      else
        xx=insert_internal(nodes[p],p,middle_element,dd_mid,ind);     // dd is NULL means we won't insert anything

      ind+=xx;
      cout<<"Leaf with parent non-null:- ind+=xx: "<<ind<<"; p: "<<p<<endl;


      cout<<endl;
      p=parent(ind);
      int ind2=prefix_sum[p];
      cout<<"prefix_sum: "<<endl;
      for(int i=0;i<k2 && prefix_sum[i]!=MAX;i++)
      {
        cout<<prefix_sum[i]<<" ";
      }
      cout<<endl;

      for(int i=0;i<k1;i++)
      {
        if(nodes[i]!=NULL)
        {
          cout<<"nodes["<<i<<"]->key: "<<endl;
          for(int j=0;j<nk;j++)
          {
            cout<<(nodes[i]->key)[j]<<" ";
          }
          cout<<endl;
          cout<<"nodes["<<i<<"]->data: "<<endl;
          for(int j=0;j<nk;j++)
          {
            cout<<"data["<<j<<"]- "<<endl;
            for(int f=0;f<m;f++)
            {
              cout<<(nodes[i]->data)[j][f]<<" ";
            }
            cout<<endl;
          }
          cout<<endl;
        }
      }

      cout<<"p: "<<p<<" ; ind: "<<ind<<"; ind2: "<<ind2<<endl;

      cout<<"prefix_sum in full leaf whose parent is not null, before: "<<endl;
      for(int i=0;i<k2 && prefix_sum[i]!=MAX;i++)
      {
        cout<<prefix_sum[i]<<" ";
      }
      cout<<endl;

      //if(flag==false)
      //{
      cout<<"incremented_index: "<<incremented_index<<endl;
        for(int i=0;i<k2;i++)
        {
          if(prefix_sum[i]==MAX)
            break;

          if(prefix_sum[i]>ind2)
          {
            cout<<"flag: "<<flag<<endl;
            if(!(prefix_sum[i]==incremented_index && flag==true))
              prefix_sum[i]+=1;
          }
        }
      //}

      cout<<"prefix_sum in full leaf whose parent is not null,after : "<<endl;
      for(int i=0;i<k2 && prefix_sum[i]!=MAX;i++)
      {
        cout<<prefix_sum[i]<<" ";
      }
      cout<<endl;

      return xx+1;
    }
    else                                                    // non-leaf
    {
      int x=0; int no_of_keys_of_v1=0; int xx=0;
      for(int i=0;i<nk;i++)
      {
        if(nn->key[i]!=MAX)
          no_of_keys_of_v1++;
        else
            break;
      }

      int p=parent(ind);
      if(p==-1 || nodes[p]==NULL)                              // parent is NULL
      {
        cout<<endl;
        cout<<"In non-leaf with parent NULL, ind: "<<ind<<"; parent(ind): "<<p<<endl;
        put_in_array(prefix_sum,k2,1,0);
        x=2;

        node* nn3; nn3=(node*)malloc(sizeof(node));
        init(nn3);
        //cout<<"middle_element: "<<middle_element<<endl;
        put_in_middle(nn3,0,middle_element,dd_mid);
        //cout<<"middle_element: "<<middle_element<<endl;
        put_in_nodes(0,nn3);

        int pf_2=-1;
        int ss=prefix_sum[1]+2;
        cout<<"ss: "<<ss<<" ; orig+2: "<<orig+2<<"; ind: "<<ind<<endl;
        if(ss>orig+2)
        {
          pf_2=ss-2+no_of_keys_of_v1;
          flag=false;
        }
        else
        {
          pf_2=ss-2+no_of_keys_of_v1+1;
          flag=true;
        }

        cout<<"pf_2: "<<pf_2<<endl;
        incremented_index=pf_2+2;;
        cout<<"incremented_index in orig+2: "<<incremented_index<<endl;
        put_in_array(prefix_sum,k2,pf_2,2);

        cout<<"prefix_sum before "<<endl;
        for(int i=0;i<k2 && prefix_sum[i]!=MAX;i++)
        {
          cout<<prefix_sum[i]<<" ";
        }
        cout<<endl;

        for(int i=1;i<k2;i++)
        {
          if(prefix_sum[i]==MAX)
            break;

          if(prefix_sum[i]>ind)
            prefix_sum[i]+=x;
        }

        cout<<"prefix_sum after: "<<endl;
        for(int i=0;i<k2 && prefix_sum[i]!=MAX;i++)
        {
          cout<<prefix_sum[i]<<" ";
        }
        cout<<endl;

        return 2;
      }
      else
      {
        int p=parent(ind);
        int xx=insert_internal(nodes[p],p,middle_element,dd_mid,ind);
        ind+=xx;
        x=1;
      }
      cout<<endl;
      cout<<"In non-leaf with parent non-null, ind+=xx: "<<ind<<"; parent(ind): "<<p<<endl;

      for(int i=0;i<k2;i++)
      {
        if(prefix_sum[i]==MAX)
          break;

        if(prefix_sum[i]>ind)
          prefix_sum[i]+=x;
      }

      int pf_2=-1;
      int ss=prefix_sum[ind];
      cout<<"ss: "<<ss<<" ; orig+xx+1: "<<orig+xx+1<<"; ind: "<<ind<<"; xx: "<<xx<<"; orig: "<<orig<<endl;
      if(ss>orig+xx+1)
      {
        pf_2=ss+no_of_keys_of_v1;
        flag=false;
      }
      else
      {
        pf_2=ss+no_of_keys_of_v1+1;
        flag=true;
      }

      cout<<"pf_2: "<<pf_2<<endl;
      incremented_index=pf_2;
      cout<<"incremented_index in orig+ind: "<<incremented_index<<endl;
      put_in_array(prefix_sum,k2,pf_2,ind+1);

      return (xx+1);
    }

  }
}

void insert(int* dd)               // insert key and data dd into nodes and update prefix_sum
{
  int key=dd[0];
  int ind=search(key);             // gives index of leaf in nodes[], where key can be inserted
  node* nn=nodes[ind];

  if(nn==NULL)                      // nodes[0] is NULL
  {
    nn=(node*)malloc(sizeof(node));
    init(nn);
    nn->key[0]=key;
    for(int j=0;j<m;j++)
    {
      nn->data[0][j]=dd[j];
    }

    nodes[0]=nn;

  }
  else                            // we have the leaf node nn to which original function can be applied
  {
    insert_internal(nn,ind,key,dd,-1);
  }

  cout<<"prefix_sum: "<<endl;
  for(int i=0;i<k2 && prefix_sum[i]!=MAX;i++)
  {
    cout<<prefix_sum[i]<<" ";
  }
  cout<<endl;

  cout<<"After inserting key "<<dd[0]<<": "<<endl;
  for(int i=0;i<k1;i++)
  {
    if(nodes[i]!=NULL)
    {
      cout<<"nodes["<<i<<"]->key: "<<endl;
      for(int j=0;j<nk;j++)
      {
        cout<<(nodes[i]->key)[j]<<" ";
      }
      cout<<endl;
      cout<<"nodes["<<i<<"]->data: "<<endl;
      for(int j=0;j<nk;j++)
      {
        cout<<"data["<<j<<"]- "<<endl;
        for(int f=0;f<m;f++)
        {
          cout<<(nodes[i]->data)[j][f]<<" ";
        }
        cout<<endl;
      }
      cout<<endl;
    }
  }

}


int getMax(int* arr, int n)      // get max value in array
{
  cout<<"In get_max"<<endl;
  cout<<"n: "<<n<<endl;
  for(int i=0;i<n;i++)
    cout<<arr[i]<<" ";
  cout<<endl;
	int mx = arr[0];
	for (int i = 1; i < n; i++)
		if (arr[i] > mx)
			mx = arr[i];
	return mx;
}

// A function to do counting sort of arr[] according to
// the digit represented by exp.
void countSort(int arr[], int n, int exp)
{
	int output[n]; // output array
	int i, count[10] = { 0 };

	// Store count of occurrences in count[]
	for (i = 0; i < n; i++)
		count[(arr[i] / exp) % 10]++;

	// Change count[i] so that count[i] now contains actual
	// position of this digit in output[]
	for (i = 1; i < 10; i++)
		count[i] += count[i - 1];

	// Build the output array
	for (i = n - 1; i >= 0; i--) {
		output[count[(arr[i] / exp) % 10] - 1] = arr[i];
		count[(arr[i] / exp) % 10]--;
	}

	// Copy the output array to arr[], so that arr[] now
	// contains sorted numbers according to current digit
	for (i = 0; i < n; i++)
		arr[i] = output[i];
}

void radixsort(int* arr, int d, int N)          // only sort based on Nth bit from LSB to MSB
{
  cout<<"d: "<<d<<endl;
	// Find the maximum number to know number of digits
	int m = getMax(arr, d);
  printf("m: %d; N:%d\n",m,N );
  //cout<<"m: "<<m<<" ; N: "<<N<<endl;

	for (int exp = N; m / exp > 0; exp *= 10)
		countSort(arr, d, exp);
}

void pre_sort(int* c, int d)
{
	int B=7; int K=16;
  cout<<"In pre_sort"<<endl;
  int T=0;
  for(T=0;T<k1;T++)
  {
    if(nodes[T]==NULL)
      break;
  }
 avg_tree_size+=T;     // T is number of nodes present
 T_count++;

  cout<<"size of tree T: "<<T<<endl;
	int N=B-floor(log2(2^B/(T)*K));
  cout<<"N: "<<N<<endl;
	radixsort(c,d,N);
}

__global__ void srch(int* c, int* ll, int* query_index_arr,int* max_comp, int* d, int* qq)  // GS=3, no_of_queries_in_warp=6
{
	int id=blockIdx.x*blockDim.x+threadIdx.x; int q=*qq;
  printf("*ll: %d\n",*ll);
  printf("nk_gpu: %d\n",nk_gpu);
  int ii=(id/32)*no_of_queries_in_warp+(id%32)/GS;
  if(id==0)
  {
    printf("c:- \n");
    for(int i=0;i<*d;i++)
    {
      printf("%d ",c[i]);
    }
    printf("\n");
  }

	if(id<*ll && id%32<no_of_queries_in_warp*GS && ii<*d)
	{
		int cmp=0;                      // 3 id's processing 1 query
		int key=c[ii];                                      // id of same query get same key
		//int xx;
		int i=0; //node* ptr; ptr=nodes[0];
	  int j=0;
    printf("id: %d; ii: %d\n",id,ii);
    if(id==3)
    {
      printf("prefix_sum_gpu: \n");
      for(int i=0;i<k2_gpu;i++)
      {
        printf("%d ",prefix_sum_gpu[i]);
      }
      printf("\n");
    }

	  while(true)
	  {
      for(int r=0;r<q;r++)
        query_index_arr[r]=-1;
	    //ptr=nodes[i];
      printf("In i: %d\n",i);
	    j=i;
	    if(nodes_gpu[i]==NULL)                     // no node inserted yet
	      return;
	    int* key_arr=nodes_gpu[i]->key;  int ke_id=-1;

        printf("key_arr for id=%d: \n",id);
        for(int r=0;r<nk_gpu;r++)
        {
          printf("key_arr[%d]- %d ",r,key_arr[r]);
        }
        printf("\n");


			if(key_arr[0]>key)
			{
        printf("1st- id: %d, key: %d, ii:%d\n",id,key,ii);
				ke_id=0; query_index_arr[ii]=0;
        cmp+=1;
			}
			else if(key_arr[nk_gpu-1]<key)
			{
        printf("2nd- id: %d, key: %d, ii:%d\n",id,key,ii);
				ke_id=nk_gpu; query_index_arr[ii]=nk_gpu;
        cmp+=1;
			}
			else
			{
        printf("3rd- id: %d, key: %d, ii:%d\n",id,key,ii);

				for(int k=id%GS;k<nk_gpu && query_index_arr[ii]==-1 ;k+=GS)
				{
          printf("id: %d, k:%d, ii:%d\n",id,k,ii);
          cmp+=GS;
					if(key_arr[k]==key)
					{
						ke_id=k;
						query_index_arr[ii]=ke_id;
            //if(prefix_sum_gpu[prefix_sum_gpu[i]+k]==MAX)
            if(prefix_sum_gpu[i]==MAX)
              printf("%d and %d are EQUAL!!!!!\n",key,key_arr[k]);
						break;
					}
					else if(k+1<nk_gpu && key_arr[k]<key && key_arr[k+1]>key)
					{
						ke_id=k+1;
						query_index_arr[ii]=ke_id;
            printf("In id:%d ;ke_id: %d\n",id,ke_id);
						break;
					}
          printf("id:%d ;ke_id: %d\n",id,ke_id);
				}
			}

      printf("query_index_arr[%d]= %d\n",ii,query_index_arr[ii]);
			int ind=query_index_arr[ii];       // ind,i same for all threads of query
	    //int ind=lb(key_arr,nk,key);

      printf("i: %d; ind: %d; prefix_sum_gpu[i]: %d\n",i,ind,prefix_sum_gpu[i]);
	    if(prefix_sum_gpu[i]==MAX)                  // reached a leaf
	      break;
	    i=prefix_sum_gpu[i]+ind;
	    if(key_arr[ind]==key)
	      i++;

      printf("Next going to i=%d\n",i);
      printf("\n");
                        			// threads that change a particular query_index_arr[ii] belong to same warp. So, barriers not needed anywhere.
	  }

	  int* key_arr=nodes_gpu[j]->key; query_index_arr[ii]=-1; int ke_id=-1;

		for(int k=id%GS; k<nk_gpu && query_index_arr[ii]==-1 ;k+=GS)
		{
			if(key_arr[k]==key)
			{
				ke_id=k;
				query_index_arr[ii]=ke_id;
				break;
			}
		}
	  //cout<<"SEARCH"<<endl;
	  if(key_arr[query_index_arr[ii]]==key)
	  {
	    int** dd=nodes_gpu[j]->data;
	    for(int r=0;r<m_gpu;r++)
	        printf("%d\n",dd[query_index_arr[ii]][r]);
	    printf("\n");
	  }

		atomicMax(max_comp,cmp);    // among all queries, max_comp is # of comparison steps of the querry that makes the warp do maximum comparisons
		printf("max_comp: %d\n",*max_comp);
	}
}

__global__ void init_pref_sum(int* pref)                 // Initialise prefix_sum_gpu using pref
{
  prefix_sum_gpu=(int*)malloc(sizeof(int)*k2_gpu);
  for(int i=0;i<k2_gpu;i++)
  {
    prefix_sum_gpu[i]=pref[i];
  }
}

void update()                                          // copies nodes[] and prefix_sum from CPU to GPU
{
  //int D=max_diff(b,q);
  int* pref;
  hipMalloc(&pref,sizeof(int)*k2);
  hipMemcpy(pref,prefix_sum,sizeof(int)*k2,hipMemcpyHostToDevice);
  init_pref_sum<<<1,1>>>(pref);

  iii<<<1,1>>>();
  hipDeviceSynchronize();

  printf("nodes_gpu initialized\n");

 for(int i=0;i<k1 && nodes[i]!=NULL;i++)
 {
     int* gpu_key; int* i2;  hipMalloc(&gpu_key,sizeof(int)*nk); hipMalloc(&i2,sizeof(int));
     hipMemcpy(gpu_key,nodes[i]->key,sizeof(int)*nk,hipMemcpyHostToDevice);
     hipMemcpy(i2,&i,sizeof(int),hipMemcpyHostToDevice);

     cpy_key<<<1,1>>>(gpu_key,i2);
     hipDeviceSynchronize();
     cout<<"Key "<<i<<" copied"<<endl;

     for(int ii=0;ii<nk;ii++)
     {
       int* ii2;  hipMalloc(&ii2,sizeof(int)); hipMemcpy(ii2,&ii,sizeof(int),hipMemcpyHostToDevice);

       //int* j2; hipMalloc(&j2,sizeof(int)); hipMemcpy(j2,&j,sizeof(int),hipMemcpyHostToDevice);
       int* gpu_data; hipMalloc(&gpu_data,sizeof(int)*m);
       hipMemcpy(gpu_data,(nodes[i]->data)[ii],sizeof(int)*m,hipMemcpyHostToDevice);

       cpy_data<<<1,1>>>(gpu_data,i2,ii2);
       hipDeviceSynchronize();
     }

     cout<<"Data "<<i<<" copied"<<endl;

 }

  cout<<"Printing nodes_gpu on GPU"<<endl;
  print<<<1,1>>>();
  hipDeviceSynchronize();
}


void func(int** a, int *b, int q)                                           // a has data of queries; b has index of inserts in a
{
  cout<<"b:-"<<endl;
  for(int i=0;i<q;i++)
  {
    cout<<b[i]<<" ";
  }
  cout<<endl; int vald=0;

	for(int i=0;i<q-1 && b[i]!=MAX && b[i+1]!=MAX;i++)         // 1 index of a has the element to be inserted. 0th has the number 2. b[i] is the index in a, where node to be inserted is present
	{
    vald++;                     // vald=len(b)-1
		int d=b[i+1]-b[i]-1;        // d queries
    cout<<"i: "<<i<<" ; q: "<<q<<endl;
    cout<<"b[i]: "<<b[i]<<" ; b[i+1]: "<<b[i+1]<<endl;
                                               // d queries done in paallel
    int* aa; aa=(int*)malloc(sizeof(int)*m);  cout<<"Inserting"<<endl;

    for(int j=1;j<=m;j++)
    {
      aa[j-1]=a[b[i]][j];
      cout<<aa[j-1]<<" ";
    }
    cout<<endl;

    cout<<"Inserting b["<<i<<"]"<<endl;
    flag=false;
		insert(aa);
    update();                           // call update everytime after insert to copy nodes to nodes_gpu


    cout<<"Back from insert"<<endl;
    int* c; c=(int*)malloc(sizeof(int)*d); int cnt=0;

    for(int j=b[i]+1;j<b[i+1];j++)                              // int* c=[a[b[i]] to a[b[i+1]]];
    {
      c[j-b[i]-1]=a[j][1]; cnt++;
    }
    cout<<"c: "<<endl;
    for(int j=0;j<d;j++)
    {
      cout<<c[j]<<" ";
    }
    cout<<endl;
    cout<<"cnt: "<<cnt<<endl;


    if(cnt>0)
    {
  		pre_sort(c,d);
      cout<<"c after pre-sorting: "<<endl;
      for(int j=0;j<d;j++)
      {
        cout<<c[j]<<" ";
      }
      cout<<endl;


  		int* query_index_arr; int* q_cpu;   int* max_comp;int* yy;   int* ll_cpu; int* ll_gpu;   int* c_gpu;  int* max_cmp_cpu;   int* d_gpu; int* qq;
  		q_cpu=(int*)malloc(sizeof(int)*d); yy=(int*)malloc(sizeof(int)); ll_cpu=(int*)malloc(sizeof(int)); max_cmp_cpu=(int*)malloc(sizeof(int));
  		for(int j=0;j<d;j++)
  			q_cpu[j]=-1;

  		*yy=0; *ll_cpu=ceil(d*1.0/no_of_queries_in_warp_cpu)*32;   // ll threads required for the d queries

  		hipMalloc(&query_index_arr,sizeof(int)*d); hipMalloc(&max_comp,sizeof(int));
  		hipMemcpy(query_index_arr,q_cpu,sizeof(int)*d,hipMemcpyHostToDevice);

  		hipMalloc(&max_comp,sizeof(int));
  		hipMemcpy(max_comp,yy,sizeof(int),hipMemcpyHostToDevice);

      cout<<"*ll_cpu: "<<*ll_cpu<<endl;
  		hipMalloc(&ll_gpu,sizeof(int));
  		hipMemcpy(ll_gpu,ll_cpu,sizeof(int),hipMemcpyHostToDevice);

      hipMalloc(&d_gpu,sizeof(int));
      hipMemcpy(d_gpu,&d,sizeof(int),hipMemcpyHostToDevice);

      hipMalloc(&qq,sizeof(int));
      hipMemcpy(qq,&q,sizeof(int),hipMemcpyHostToDevice);

      hipMalloc(&c_gpu,sizeof(int)*d);
  		hipMemcpy(c_gpu,c,sizeof(int)*d,hipMemcpyHostToDevice);

      cout<<"Before srch call"<<endl;
      cout<<"ceil(d*1.0/no_of_queries_in_warp_cpu): "<<ceil(d*1.0/no_of_queries_in_warp_cpu)<<endl;

  		srch<<<ceil(d*1.0/no_of_queries_in_warp_cpu),32>>>(c_gpu,ll_gpu,query_index_arr,max_comp,d_gpu,qq);   // max_comp is count of max_comparisons made in this search
  		hipDeviceSynchronize();                                    // 1 block serves no_of_queries_in_warp queries
      cout<<"After srch"<<endl;
  		hipMemcpy(max_cmp_cpu,max_comp,sizeof(int),hipMemcpyDeviceToHost);

  		int product=(GS_cpu)*(*max_cmp_cpu);
      sum_of_den+=product;
      printf("max_cmp_cpu: %d\n",*max_cmp_cpu);
    }
	}

  int* aa; aa=(int*)malloc(sizeof(int)*m);
  cout<<"Inserting"<<endl;
  for(int j=1;j<=m;j++)
  {
    aa[j-1]=a[b[vald]][j];
    cout<<aa[j-1]<<" ";
  }
  cout<<endl;

  cout<<"Inserting b["<<vald<<"]"<<endl;
  flag=false;
  insert(aa);
  update();

  cout<<"vald: "<<vald<<endl; cout<<"b[vald]: "<<b[vald]<<"; b[vald+1]: "<<b[vald+1]<<endl;
  cout<<"q: "<<q<<endl;
  if(b[vald]<q)
  {
    int d=q-b[vald]-1;
    cout<<"d: "<<d<<endl;
    int* c; c=(int*)malloc(sizeof(int)*d); int cnt=0; int i=vald;

    for(int j=b[i]+1;j<q;j++)                              // int* c=[a[b[i]] to a[b[i+1]]];
    {
      c[j-b[i]-1]=a[j][1]; cnt++;
    }
    cout<<"c: "<<endl;
    for(int j=0;j<d;j++)
    {
      cout<<c[j]<<" ";
    }
    cout<<endl;
    cout<<"cnt: "<<cnt<<endl;

    if(cnt>0)
    {
      cout<<"Before pre_sorting"<<endl;
  		pre_sort(c,d);
      cout<<"c after pre-sorting: "<<endl;
      for(int j=0;j<d;j++)
      {
        cout<<c[j]<<" ";
      }
      cout<<endl;

  		int* query_index_arr; int* q_cpu;   int* max_comp;int* yy;   int* ll_cpu; int* ll_gpu;   int* c_gpu;  int* max_cmp_cpu;int* d_gpu;  int* qq;
  		q_cpu=(int*)malloc(sizeof(int)*d); yy=(int*)malloc(sizeof(int)); ll_cpu=(int*)malloc(sizeof(int)); max_cmp_cpu=(int*)malloc(sizeof(int));
  		for(int j=0;j<d;j++)
  			q_cpu[j]=-1;

  		*yy=0; *ll_cpu=ceil(d*1.0/no_of_queries_in_warp_cpu)*32;

  		hipMalloc(&query_index_arr,sizeof(int)*d); hipMalloc(&max_comp,sizeof(int));
  		hipMemcpy(query_index_arr,q_cpu,sizeof(int)*d,hipMemcpyHostToDevice);

  		hipMalloc(&max_comp,sizeof(int));
  		hipMemcpy(max_comp,yy,sizeof(int),hipMemcpyHostToDevice);

      cout<<"*ll_cpu: "<<*ll_cpu<<endl;
  		hipMalloc(&ll_gpu,sizeof(int));
  		hipMemcpy(ll_gpu,ll_cpu,sizeof(int),hipMemcpyHostToDevice);

      hipMalloc(&qq,sizeof(int));
      hipMemcpy(qq,&q,sizeof(int),hipMemcpyHostToDevice);

      hipMalloc(&d_gpu,sizeof(int));
      hipMemcpy(d_gpu,&d,sizeof(int),hipMemcpyHostToDevice);

      hipMalloc(&c_gpu,sizeof(int)*d);
  		hipMemcpy(c_gpu,c,sizeof(int)*d,hipMemcpyHostToDevice);

      cout<<"Before srch call"<<endl;
      cout<<"ceil(d*1.0/no_of_queries_in_warp_cpu): "<<ceil(d*1.0/no_of_queries_in_warp_cpu)<<endl;
      cout<<"d: "<<d<<" ; no_of_queries_in_warp_cpu: "<<no_of_queries_in_warp_cpu<<endl;

  		srch<<<ceil(d*1.0/no_of_queries_in_warp_cpu),32>>>(c_gpu,ll_gpu,query_index_arr,max_comp,d_gpu,qq);
  		hipDeviceSynchronize();                                    // 1 block serves no_of_queries_in_warp queries
      cout<<"After srch"<<endl;
  		hipMemcpy(max_cmp_cpu,max_comp,sizeof(int),hipMemcpyDeviceToHost);

  		int product=(GS_cpu)*(*max_cmp_cpu);
      sum_of_den+=product;
      printf("max_cmp_cpu: %d\n",*max_cmp_cpu);

    }
  }

}

int main()
{

  nodes=(node**)malloc(sizeof(node*)*k1);
  prefix_sum=(int*)malloc(sizeof(int)*k2);

  for(int i=0;i<k1;i++)
  {
    nodes[i]=NULL;
    if(i<k2)
      prefix_sum[i]=MAX;
  }

  int q;
  cout<<"Enter number of queries:- "<<endl;
  cin>> q;  // no_of_queries
  int** a; int *b; b=(int*)malloc(sizeof(int)*q);
  a=(int**)malloc(sizeof(int*)*q);
  for(int i=0;i<q;i++)
  {
    a[i]=(int*)malloc(sizeof(int)*(m+1));
  }
  //int a[q][m_cpu+1]; int b[q];
  for(int i=0;i<q;i++)
  {
    b[i]=MAX;
  }
  for(int i=0;i<q;i++)
  {
    for(int j=0;j<=m;j++)
      a[i][j]=MAX;
  }

  for(int i=0;i<q;i++)
  {
    cout<<"Enter 1 for inserting a tuple"<<endl;
    cout<<"Enter 2 for searching for a key"<<endl;
    cout<<"Enter 3 for doing range_query"<<endl;
    int kk; cin>>kk;
    if(kk==1)                      // insert
    {
      int dd[m]; a[i][0]=1;
      cout<<"Enter "<<m<<" elements of a tuple, of which 1st should be key:- "<<endl;
      for(int j=0;j<m;j++)
      {
        cin>>dd[j];
        a[i][j+1]=dd[j];
      }
      flag=false;
      //insert(dd);
    }
    else if(kk==2)
    {
      cout<<"Enter key to be searched"<<endl; int ke; cin>>ke;
      a[i][0]=2; a[i][1]=ke;
      //search(ke);
    }
    // else if(kk==3)
    // {
    //   cout<<"Enter 2 keys between which range_query is done"<<endl;
    //   int k1, k2; cin>>k1>>k2;
    //   range_query(k1,k2);
    // }
  }

  int jj=0;
  for(int i=0;i<q;i++)
  {
    if(a[i][0]==1)                      // b has inserts
    {
      b[jj]=i; jj++;
    }
  }

  func(a,b,q);
  if(T_count!=0)
    avg_tree_size/=T_count;

  float throughput=32*1.0/sum_of_den;
  cout<<"nk_gpu: "<<nk<<"; GS: "<<GS_cpu<<"; no_of_queries_in_warp: "<<no_of_queries_in_warp_cpu<<";  avg_tree_size: "<<avg_tree_size<<"; throughput: "<<throughput<<endl;
  //printf("throughput: %f\n",throughput);

  return 0;
}
